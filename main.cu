#include "hip/hip_runtime.h"
#include <png_image.h>
#include <ctime>
#include "cuda_check.h"
#include "camera.h"
#include "hittable_list.h"
#include "sphere.h"
#include "material.h"

__device__ color ray_color(const ray& r, hittable_list *world, int depth, hiprandState *randState) {
    hit_record rec;
    ray curRay = r;
    //printf("in ray color\n");
    color attenuation(1, 1, 1);
    while(depth-- > 0){
        if (world->hit(curRay, 0.001f, globalvar::kInfinityGPU, rec)) {
            color nextAttenuation;
            if (rec.mat_ptr->scatter(curRay, rec, nextAttenuation, curRay, randState))
                attenuation *= nextAttenuation;
            else attenuation = vec3();
        }
        else{
            vec3 unit_direction = vectorgpu::normalize(curRay.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            return ((1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0)) * attenuation;
        }
    }
    return {};
}

__global__ void generateWorld(hittable_list *world){
    world->initObj(4);
    auto material_ground = new lambertian(color(0.8, 0.8, 0.0));
    auto material_center = new lambertian(color(0.7, 0.3, 0.3));
    auto material_left = new metal(color(0.8, 0.8, 0.8), 0);
    auto material_right   = new metal(color(0.8, 0.6, 0.2), 1);
    world->objects[0] = new sphere(point3( 0.0, -100.5, -1.0), 100.0, material_ground);
    world->objects[1] = new sphere(point3( 0.0, 0.0, -1.0),   0.5, material_center);
    world->objects[2] = new sphere(point3(-1.0, 0.0, -1.0),   0.5, material_left);
    world->objects[3] = new sphere(point3( 1.0, 0.0, -1.0),   0.5, material_right);
}

__global__ void clearWorld(hittable_list *world){
    delete world;
}

__global__ void initRandom(hiprandState *randState, int maxWidth, int maxHeight, int seed){
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    unsigned curPixel = row * maxWidth + col;
    hiprand_init(seed, curPixel, 0, &randState[curPixel]);
}

__global__ void render(vec3 *frameBuffer, int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       hittable_list *world, hiprandState *randState){
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight, sppInv = 1.0f / spp;

    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        ray r = myCamera->get_ray(u, v);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        frameBuffer[curPixel] += ray_color(r, world, maxDepth, &randState[curPixel]);
    }
    float r = sqrtf(frameBuffer[curPixel].r() * sppInv);
    float g = sqrtf(frameBuffer[curPixel].g() * sppInv);
    float b = sqrtf(frameBuffer[curPixel].b() * sppInv);
    frameBuffer[curPixel] = vec3(r, g, b);
}


int main()
{
    PngImage png(globalvar::kFrameWidth, globalvar::kFrameHeight);

    size_t frameBufferSize = globalvar::kFrameHeight * globalvar::kFrameWidth * sizeof(vec3);
    vec3 *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));

    hittable_list *world;
    checkCudaErrors(hipMalloc((void **)&world, sizeof(hittable_list)));

    camera *devCamera, *hostCamera = new camera();
    checkCudaErrors(hipMalloc((void **)&devCamera, sizeof(camera)));
    checkCudaErrors(hipMemcpy(devCamera, hostCamera, sizeof(camera), hipMemcpyHostToDevice));

    generateWorld<<<1, 1>>>(world);

    dim3 blocks(globalvar::kBlockX, globalvar::kBlockY);
    dim3 threads(globalvar::kThreadX, globalvar::kThreadY);
    //printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);

    hiprandState *devStates;
    checkCudaErrors(hipMalloc((void **)&devStates, frameBufferSize * sizeof(hiprandState)));
    srand(time(nullptr));
    int seed = rand();
    initRandom<<<blocks, threads>>>(devStates, globalvar::kFrameWidth, globalvar::kFrameHeight, seed);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    printf("Start rendering!\n");
    std::clock_t start = std::clock();
    render<<<blocks, threads>>>(frameBuffer, globalvar::kFrameWidth, globalvar::kFrameHeight,
                                globalvar::kSpp, globalvar::kMaxDepth,
                                devCamera, world, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Time Cost: "<< duration <<'\n';
    for(int row = 0; row < globalvar::kFrameHeight; row++){
        for(int col = 0; col < globalvar::kFrameWidth; col++){
            int curPixel = row * globalvar::kFrameWidth + col;
            //std::cerr << frameBuffer[curPixel] << '\n';
            png.saveColor(frameBuffer[curPixel], globalvar::kFrameHeight - row - 1, col);
        }
    }

    png.write("../output/10.png");
    clearWorld<<<1, 1>>>(world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(frameBuffer));
    checkCudaErrors(hipFree(world));
    delete []frameBuffer;
    return 0;
}

