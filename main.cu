#include "hip/hip_runtime.h"
#include <png_image.h>
#include <ctime>
#include "cuda_check.h"
#include "camera.h"
#include "render_manager.h"
#include "material.h"
#include "bvh.h"
#include "cuda2gl.h"

surface<void,cudaSurfaceType2D> surf;
camera *devCamera, *hostCamera;
RenderManager *world;
size_t frameBufferSize = globalvar::kFrameHeight * globalvar::kFrameWidth * sizeof(vec3);
hiprandState *devStates;
dim3 blocks(globalvar::kBlockX, globalvar::kBlockY);
dim3 threads(globalvar::kThreadX, globalvar::kThreadY);
double deltaTime = 0;

__device__ color ray_color(const Ray& r, RenderManager *world, int depth, hiprandState *randState) {
    hit_record rec;
    Ray curRay = r;
    //printf("in ray color\n");
    color attenuation(1, 1, 1);
    while(depth-- > 0){
        if (world->hit(curRay, 0.001f, globalvar::kInfinityGPU, rec)) {
            color nextAttenuation;
            if (world->mats[rec.matID].scatter(curRay, rec, nextAttenuation, curRay, randState))
                attenuation *= nextAttenuation;
            else attenuation = vec3();
        }
        else{
            vec3 unit_direction = vectorgpu::normalize(curRay.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            return ((1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0)) * attenuation;
        }
    }
    return {};
}

__global__ void generateWorld(RenderManager *world){
    world->initObj(5);
    world->initMat(4);

    auto material_ground = new material(color(0.8, 0.8, 0.0));
    world->addMat(material_ground);
    auto material_center = new material(color(0.1, 0.2, 0.5));
    world->addMat(material_center);
    auto material_left = new material(1.5f);
    world->addMat(material_left);
    auto material_right = new material(color(0.8, 0.6, 0.2), 1);
    world->addMat(material_right);

    world->addObj(new CudaObj(point3(0.0, -100.5, -1.0), 100.0, 0));
    world->addObj(new CudaObj(point3(0.0, 0.0, -1.0), 0.5, 1));
    world->addObj(new CudaObj(point3(-1.0, 0.0, -1.0), 0.5, 2));
    world->addObj(new CudaObj(point3(-1.0, 0.0, -1.0), -0.4, 2));
    world->addObj(new CudaObj(point3(1.0, 0.0, -1.0), 0.5, 3));
}

__global__ void generateRandomWorld(RenderManager *world, hiprandState* randState){
    int sampleNum = 0;
    int objSz = sampleNum * sampleNum * 4 + 4 + 1;
    world->initObj(objSz);
    world->initMat(objSz);

    auto ground_material = new material(color(0.5, 0.5, 0.5));
    world->addObj(new CudaObj(point3(0, -1000, 0), 1000, 0));
    world->addMat(ground_material);

    for(int i = -sampleNum; i < sampleNum; i++){
        for(int j = -sampleNum; j < sampleNum; j++){
            float choose_mat = hiprand_uniform(randState);
            point3 center(i + 0.9f * hiprand_uniform(randState), 0.2, j + 0.9f * hiprand_uniform(randState));
            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material *sphere_material;
                auto rand1 = vec3(hiprand_uniform(randState), hiprand_uniform(randState), hiprand_uniform(randState));
                auto rand2 = vec3(hiprand_uniform(randState), hiprand_uniform(randState), hiprand_uniform(randState));
                if(choose_mat < 0.8){
                    auto albedo = rand1 * rand2;
                    sphere_material = new material(albedo);
                    auto center2 = center + vec3(0, rand2.y() * 0.5f, 0);
                    world->addObj(new CudaObj(center, 0.2, world->matLastIdx));
                }
                else if(choose_mat < 0.95){
                    auto albedo = rand1 / 2 + vec3(0.5f, 0.5f, 0.5f);
                    float fuzz = rand2.x() / 2;
                    sphere_material = new material(albedo, fuzz);
                    world->addObj(new CudaObj(center, 0.2, world->matLastIdx));
                }
                else{
                    sphere_material = new material(1.5f);
                    world->addObj(new CudaObj(center, 0.2, world->matLastIdx));
                }
                world->addMat(sphere_material);

            }
        }
    }
    auto material1 = new material(1.5f);

    world->addObj(new CudaObj(point3(4, 1, 0), 1.0, world->matLastIdx));
    world->addObj(new CudaObj(point3(4, 1, 0), -0.9, world->matLastIdx));
    world->addMat(material1);

    auto material2 = new material(color(1, 0, 0.4));
    world->addObj(new CudaObj(point3(-4, 1, 0), 1.0, world->matLastIdx));
    world->addMat(material2);

    auto material3 = new material(color(0.7, 0.6, 0.5), 0.0);
    world->addObj(new CudaObj(point3(0, 1, 0), 1.0, world->matLastIdx));
    world->addMat(material3);

    printf("%f", world->mWorldBoundingBox.getMin().x());
}

__global__ void clearWorld(RenderManager *world){
    delete world;
}

__global__ void initRandom(hiprandState *randState, int maxWidth, int maxHeight, int seed){
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    unsigned curPixel = row * maxWidth + col;
    hiprand_init(seed, curPixel, 0, &randState[curPixel]);
}

__global__ void render(vec3 *frameBuffer, int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       RenderManager *world, hiprandState *randState){
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight, sppInv = 1.0f / spp;
    color finalColor = color();
    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        Ray r = myCamera->get_ray(u, v, randState);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        finalColor += ray_color(r, world, maxDepth, &randState[curPixel]);
    }
    float r = sqrtf(finalColor.r() * sppInv);
    float g = sqrtf(finalColor.g() * sppInv);
    float b = sqrtf(finalColor.b() * sppInv);
    frameBuffer[curPixel] = vec3(r, g, b);
}

union pxl_rgbx_24
{
    uint1 b32;
    struct {
        unsigned  r  : 8;
        unsigned  g  : 8;
        unsigned  b  : 8;
        unsigned  na : 8;
    };
};


bool goRender = true;

__global__ void renderBySurface(int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       RenderManager *world, hiprandState *randState, bool goRender){
    if(!goRender) return;
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight, sppInv = 1.0f / spp;

    vec3 color = vec3();

    union pxl_rgbx_24 rgbx;

    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        Ray r = myCamera->get_ray(u, v, randState);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        color += ray_color(r, world, maxDepth, &randState[curPixel]);
    }
    rgbx.r = sqrtf(color.r() * sppInv) * 255;
    rgbx.g = sqrtf(color.g() * sppInv) * 255;
    rgbx.b = sqrtf(color.b() * sppInv) * 255;
    rgbx.na = 255;

    surf2Dwrite(rgbx.b32,
                surf,
                col * sizeof(rgbx),
                row,
                hipBoundaryModeZero);
}

static void fpsCount(GLFWwindow* window)
{
    static double lastFrame  = 0.0;
    static int frameCount = 0;
    const double currentFrame = glfwGetTime();
    deltaTime = currentFrame - lastFrame;
    if (deltaTime > 0.5)
    {
        lastFrame = currentFrame;
        const double fps = (double)frameCount / deltaTime;
        int  width, height;
        char tmp[64];
        glfwGetFramebufferSize(window,&width,&height);
        sprintf_s(tmp,64,"(%u x %u) - FPS: %.2f", width, height, fps);
        glfwSetWindowTitle(window,tmp);
        frameCount = 0;
    }
    frameCount++;
}

void myGlInit(GLFWwindow** window, const int width, const int height){
    if (!glfwInit())
        exit(EXIT_FAILURE);
    glfwWindowHint(GLFW_DEPTH_BITS,            0);
    glfwWindowHint(GLFW_STENCIL_BITS,          0);

    //glfwWindowHint(GLFW_SRGB_CAPABLE,          GL_TRUE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);

    glfwWindowHint(GLFW_OPENGL_PROFILE,        GLFW_OPENGL_CORE_PROFILE);

    *window = glfwCreateWindow(width,height,"GLFW / CUDA Interop",NULL,NULL);
    if (*window == NULL)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    glfwMakeContextCurrent(*window);
    // set up GLAD
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
    // ignore vsync for now
    glfwSwapInterval(0);
}

void processInput(GLFWwindow *window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS){
        glfwSetWindowShouldClose(window, true);
        return;
    }
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        hostCamera->processKeyboard(FORWARD, deltaTime);
    else if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        hostCamera->processKeyboard(BACKWARD, deltaTime);
    else if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        hostCamera->processKeyboard(LEFT, deltaTime);
    else if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        hostCamera->processKeyboard(RIGHT, deltaTime);
    else return;
    checkCudaErrors(hipMemcpy(devCamera, hostCamera, sizeof(camera), hipMemcpyHostToDevice));
}

void initWorldStates(){
    checkCudaErrors(hipMalloc((void **)&world, sizeof(RenderManager)));
    hostCamera = new camera(
            vec3 (13,2,3),
            vec3(0,0,0), 20,
            globalvar::kAspectRatio,
            0, 10, 0.0, 1.0);
    checkCudaErrors(hipMalloc((void **)&devCamera, sizeof(camera)));
    checkCudaErrors(hipMemcpy(devCamera, hostCamera, sizeof(camera), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **)&devStates, frameBufferSize * sizeof(hiprandState)));
    srand(time(nullptr));
    int seed = rand();
    initRandom<<<blocks, threads>>>(devStates, globalvar::kFrameWidth, globalvar::kFrameHeight, seed);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    generateRandomWorld<<<1, 1>>>(world, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

void clearWorldStates(){
    clearWorld<<<1, 1>>>(world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(devStates));
    checkCudaErrors(hipFree(devCamera));
    hipDeviceReset();
}

void renderToPng(){
    PngImage png(globalvar::kFrameWidth, globalvar::kFrameHeight);
    vec3 *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));
    initWorldStates();

    printf("Start rendering!\n");
    std::clock_t start = std::clock();
    render<<<blocks, threads>>>(frameBuffer, globalvar::kFrameWidth, globalvar::kFrameHeight,
                                globalvar::kSpp, globalvar::kMaxDepth,
                                devCamera, world, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Time Cost: "<< duration <<'\n';
    for(int row = 0; row < globalvar::kFrameHeight; row++){
        for(int col = 0; col < globalvar::kFrameWidth; col++){
            int curPixel = row * globalvar::kFrameWidth + col;
            //std::cerr << frameBuffer[curPixel] << '\n';
            png.saveColor(frameBuffer[curPixel], globalvar::kFrameHeight - row - 1, col);
        }
    }
    png.write("../output2/3.png");
    checkCudaErrors(hipFree(frameBuffer));
    clearWorldStates();
}

void renderToGL(){
    initWorldStates();
    GLFWwindow *window;
    myGlInit(&window, globalvar::kFrameWidth, globalvar::kFrameHeight);

    hipStream_t stream;
    hipEvent_t  event;

    checkCudaErrors(hipStreamCreateWithFlags(&stream,hipStreamDefault));
    checkCudaErrors(hipEventCreateWithFlags(&event,hipEventBlockingSync));

    Cuda2Gl* interop = new Cuda2Gl(2);

    int width, height;
    glfwGetFramebufferSize(window,&width,&height);
    interop->updateFrameSize(width,height);
    glfwSetWindowUserPointer(window,interop);
    while (!glfwWindowShouldClose(window))
    {
        fpsCount(window);
        processInput(window);
        interop->getFrameSize(&width,&height);
        interop->mapGraphicResource(stream);
        cudaBindSurfaceToArray(surf, interop->getCudaArray());

        renderBySurface<<<blocks, threads>>>(globalvar::kFrameWidth, globalvar::kFrameHeight,
                                             globalvar::kSpp, globalvar::kMaxDepth,
                                             devCamera, world, devStates, goRender);

        interop->unMapGraphicResource(stream);
        interop->blitFramebuffer();
        interop->swapBuffer();
        glfwSwapBuffers(window);
        glfwPollEvents(); // glfwWaitEvents();
    }
    delete interop;
    glfwDestroyWindow(window);
    glfwTerminate();
    clearWorldStates();
}

int main()
{
    renderToGL();
}

