#include "hip/hip_runtime.h"
#include <png_image.h>
#include <chrono>
#include <iostream>
#include "cuda_check.h"
#include "camera.h"
#include "hittable_list.h"
#include "sphere.h"
#include "hiprand/hiprand_kernel.h"

__device__ color ray_color(const ray& r, hittable_list *world, int depth) {
    hit_record rec;
    //printf("in ray color\n");
    if (world->hit(r, 0, globalvar::kInfinityGPU, rec)) {
        return 0.5 * (rec.normal + color(1,1,1));
    }
    vec3 unit_direction = vectorgpu::normalize(r.direction());
    float t = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f - t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
}

__global__ void generateWorld(hittable_list *world){
    world->initObj(2);
    world->objects[0] = new sphere(point3(0,0,-1), 0.5);
    world->objects[1] = new sphere(point3(0,-100.5,-1), 100);
}

__global__ void clearWorld(hittable_list *world){
    delete world;
}

__global__ void initRandom(hiprandState *randState, int maxWidth, int maxHeight, int seed){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    int curPixel = row * maxWidth + col;
    hiprand_init(seed, curPixel, 0, &randState[curPixel]);
}

__global__ void render(vec3 *frameBuffer, int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       hittable_list *world, hiprandState *randState){
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    int curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight;

    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        ray r = myCamera->get_ray(u, v);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        frameBuffer[curPixel] += ray_color(r, world, maxDepth);
    }
    frameBuffer[curPixel] /= spp;
}


int main()
{
    PngImage png(globalvar::kFrameWidth, globalvar::kFrameHeight);

    size_t frameBufferSize = globalvar::kFrameHeight * globalvar::kFrameWidth * sizeof(vec3);
    vec3 *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));

    hittable_list *world;
    checkCudaErrors(hipMalloc((void **)&world, sizeof(hittable_list)));

    camera *devCamera, *hostCamera = new camera();
    checkCudaErrors(hipMalloc((void **)&devCamera, sizeof(camera)));
    checkCudaErrors(hipMemcpy(devCamera, hostCamera, sizeof(camera), hipMemcpyHostToDevice));

    generateWorld<<<1, 1>>>(world);

    dim3 blocks(kBlockX, kBlockY);
    dim3 threads(kThreadX, kThreadY);
    //printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);

    hiprandState *devStates;
    checkCudaErrors(hipMalloc((void **)&devStates, frameBufferSize * sizeof(hiprandState)));
    srand(time(nullptr));
    int seed = rand();
    initRandom<<<blocks, threads>>>(devStates, kFrameWidth, kFrameHeight, seed);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::system_clock::now();
    render<<<blocks, threads>>>(frameBuffer, kFrameWidth, kFrameHeight, kSpp, kMaxDepth,
                                devCamera, world, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::system_clock::now();
    auto duration = end - start;
    std::cout <<  "Time cost "
                   << double(duration.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den
                   << "\n";
    for(int row = 0; row < kFrameHeight; row++){
        for(int col = 0; col < kFrameWidth; col++){
            int curPixel = row * kFrameWidth + col;
            //std::cerr << frameBuffer[curPixel] << '\n';
            png.saveColor(frameBuffer[curPixel], kFrameHeight - row - 1, col);
        }
    }

    png.write("../output/8.png");
    clearWorld<<<1, 1>>>(world);
    checkCudaErrors(hipFree(frameBuffer));
    checkCudaErrors(hipFree(world));
    delete []frameBuffer;
    return 0;
}

