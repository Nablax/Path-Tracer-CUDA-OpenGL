#include "hip/hip_runtime.h"
#include <png_image.h>
#include <ctime>
#include "cuda_check.h"
#include "camera.h"
#include "render_manager.h"
#include "sphere.h"
#include "material.h"
#include "moving_sphere.h"
#include "bvh.h"
#include "cuda2gl.h"

surface<void,cudaSurfaceType2D> surf;

__device__ color ray_color(const ray& r, RenderManager *world, int depth, hiprandState *randState) {
    hit_record rec;
    ray curRay = r;
    //printf("in ray color\n");
    color attenuation(1, 1, 1);
    while(depth-- > 0){
        if (world->hit(curRay, 0.001f, globalvar::kInfinityGPU, rec)) {
            color nextAttenuation;
            if (rec.mat_ptr->scatter(curRay, rec, nextAttenuation, curRay, randState))
                attenuation *= nextAttenuation;
            else attenuation = vec3();
        }
        else{
            vec3 unit_direction = vectorgpu::normalize(curRay.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            return ((1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0)) * attenuation;
        }
    }
    return {};
}

__global__ void generateWorld(RenderManager *world){
    world->initObj(5);
    world->initMat(4);

    auto material_ground = new lambertian(color(0.8, 0.8, 0.0));
    world->addMat(material_ground);
    auto material_center = new lambertian(color(0.1, 0.2, 0.5));
    world->addMat(material_center);
    auto material_left = new dielectric(1.5f);
    world->addMat(material_left);
    auto material_right = new metal(color(0.8, 0.6, 0.2), 1);
    world->addMat(material_right);

    world->addObj(new sphere(point3( 0.0, -100.5, -1.0), 100.0, material_ground));
    world->addObj(new sphere(point3( 0.0, 0.0, -1.0),   0.5, material_center));
    world->addObj(new sphere(point3(-1.0, 0.0, -1.0),   0.5, material_left));
    world->addObj(new sphere(point3(-1.0, 0.0, -1.0),   -0.4, material_left));
    world->addObj(new sphere(point3( 1.0, 0.0, -1.0),   0.5, material_right));
}

__global__ void generateRandomWorld(RenderManager *world, hiprandState* randState){
    world->initObj(600);
    world->initMat(600);

    auto ground_material = new lambertian(color(0.5, 0.5, 0.5));
    world->addObj(new sphere(point3(0,-1000,0), 1000, ground_material));
    world->addMat(ground_material);
    int sampleNum = 0;
    for(int i = -sampleNum; i < sampleNum; i++){
        for(int j = -sampleNum; j < sampleNum; j++){
            float choose_mat = hiprand_uniform(randState);
            point3 center(i + 0.9f * hiprand_uniform(randState), 0.2, j + 0.9f * hiprand_uniform(randState));
            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                material *sphere_material;
                auto rand1 = vec3(hiprand_uniform(randState), hiprand_uniform(randState), hiprand_uniform(randState));
                auto rand2 = vec3(hiprand_uniform(randState), hiprand_uniform(randState), hiprand_uniform(randState));
                if(choose_mat < 0.8){
                    auto albedo = rand1 * rand2;
                    sphere_material = new lambertian(albedo);
                    auto center2 = center + vec3(0, rand2.y() * 0.5f, 0);
                    world->addObj(new moving_sphere(center, center2, 0.0, 1.0, 0.2, sphere_material));
                }
                else if(choose_mat < 0.95){
                    auto albedo = rand1 / 2 + vec3(0.5f, 0.5f, 0.5f);
                    float fuzz = rand2.x() / 2;
                    sphere_material = new metal(albedo, fuzz);
                    world->addObj(new sphere(center, 0.2, sphere_material));
                }
                else{
                    sphere_material = new dielectric(1.5f);
                    world->addObj(new sphere(center, 0.2, sphere_material));
                }
                world->addMat(sphere_material);

            }
        }
    }
    auto material1 = new dielectric(1.5f);
    world->addMat(material1);
    world->addObj(new sphere(point3(4, 1, 0), 1.0, material1));
    world->addObj(new sphere(point3(4, 1, 0), -0.9, material1));

    auto material2 = new lambertian(color(1, 0, 0.4));
    world->addMat(material2);
    world->addObj(new sphere(point3(-4, 1, 0), 1.0, material2));

    auto material3 = new metal(color(0.7, 0.6, 0.5), 0.0);
    world->addMat(material3);
    world->addObj(new sphere(point3(0, 1, 0), 1.0, material3));
}

__global__ void clearWorld(RenderManager *world){
    delete world;
}

__global__ void initRandom(hiprandState *randState, int maxWidth, int maxHeight, int seed){
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    unsigned curPixel = row * maxWidth + col;
    hiprand_init(seed, curPixel, 0, &randState[curPixel]);
}

__global__ void render(vec3 *frameBuffer, int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       RenderManager *world, hiprandState *randState){
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight, sppInv = 1.0f / spp;

    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        ray r = myCamera->get_ray(u, v, randState);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        frameBuffer[curPixel] += ray_color(r, world, maxDepth, &randState[curPixel]);
    }
    float r = sqrtf(frameBuffer[curPixel].r() * sppInv);
    float g = sqrtf(frameBuffer[curPixel].g() * sppInv);
    float b = sqrtf(frameBuffer[curPixel].b() * sppInv);
    frameBuffer[curPixel] = vec3(r, g, b);
}

__global__ void renderPerSpp(vec3 *frameBuffer, int maxWidth, int maxHeight, float sppInv, int maxDepth,
                       camera *myCamera,
                       RenderManager *world, hiprandState *randState){

    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;

    float u = (col + hiprand_uniform(&randState[curPixel])) / maxWidth;
    float v = (row + hiprand_uniform(&randState[curPixel])) / maxHeight;
    ray tmpR = myCamera->get_ray(u, v, randState);
    //printf("%f %f %f\n", u, v, myCamera->fl);
    vec3 retColor = ray_color(tmpR, world, maxDepth, &randState[curPixel]) * sppInv;
    frameBuffer[curPixel] += retColor;
//    for(int i = 0; i < 3; i++){
//        atomicAdd(&frameBuffer[curPixel].e[i], retColor.e[i]);
//    }
}

union pxl_rgbx_24
{
    uint1 b32;
    struct {
        unsigned  r  : 8;
        unsigned  g  : 8;
        unsigned  b  : 8;
        unsigned  na : 8;
    };
};


bool goRender = true;

__global__ void renderBySurface(int maxWidth, int maxHeight, int spp, int maxDepth,
                       camera *myCamera,
                       RenderManager *world, hiprandState *randState, bool goRender){
    if(!goRender) return;
    //printf("%d %d %d %d %d %d\n", blockDim.x, threadIdx.x, threadIdx.x, blockDim.y, threadIdx.y, threadIdx.y);
    unsigned col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned row = threadIdx.y + blockIdx.y * blockDim.y;

    if(row >= maxHeight || col >= maxWidth) return;
    //printf("%d %d %d %d\n", row, col, maxWidth, maxHeight);
    unsigned curPixel = row * maxWidth + col;
    float maxWidthInv = 1.0f / maxWidth, maxHeightInv = 1.0f / maxHeight, sppInv = 1.0f / spp;

    vec3 color = vec3();

    union pxl_rgbx_24 rgbx;

    for(int i = 0; i < spp; i++){
        float u = (col + hiprand_uniform(&randState[curPixel])) * maxWidthInv;
        float v = (row + hiprand_uniform(&randState[curPixel])) * maxHeightInv;
        ray r = myCamera->get_ray(u, v, randState);
        //printf("%f %f %f\n", u, v, myCamera->fl);
        color += ray_color(r, world, maxDepth, &randState[curPixel]);
    }
    rgbx.r = sqrtf(color.r() * sppInv) * 255;
    rgbx.g = sqrtf(color.g() * sppInv) * 255;
    rgbx.b = sqrtf(color.b() * sppInv) * 255;
    rgbx.na = 255;

    surf2Dwrite(rgbx.b32,
                surf,
                col * sizeof(rgbx),
                (maxHeight - row - 1),
                hipBoundaryModeZero);
}

static
void
pxl_glfw_fps(GLFWwindow* window)
{
    // static fps counters
    static double stamp_prev  = 0.0;
    static int    frame_count = 0;

    // locals
    const double stamp_curr = glfwGetTime();
    const double elapsed    = stamp_curr - stamp_prev;

    if (elapsed > 0.5)
    {
        stamp_prev = stamp_curr;

        const double fps = (double)frame_count / elapsed;

        int  width, height;
        char tmp[64];

        glfwGetFramebufferSize(window,&width,&height);

        sprintf_s(tmp,64,"(%u x %u) - FPS: %.2f",width,height,fps);

        glfwSetWindowTitle(window,tmp);

        frame_count = 0;
    }

    frame_count++;
}

void myGlInit(GLFWwindow** window, const int width, const int height){
    if (!glfwInit())
        exit(EXIT_FAILURE);

    glfwWindowHint(GLFW_DEPTH_BITS,            0);
    glfwWindowHint(GLFW_STENCIL_BITS,          0);

    //glfwWindowHint(GLFW_SRGB_CAPABLE,          GL_TRUE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);

    glfwWindowHint(GLFW_OPENGL_PROFILE,        GLFW_OPENGL_CORE_PROFILE);

    *window = glfwCreateWindow(width,height,"GLFW / CUDA Interop",NULL,NULL);

    if (*window == NULL)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }

    glfwMakeContextCurrent(*window);

    // set up GLAD
    gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);

    // ignore vsync for now
    glfwSwapInterval(0);
}

int main()
{
    PngImage png(globalvar::kFrameWidth, globalvar::kFrameHeight);

    size_t frameBufferSize = globalvar::kFrameHeight * globalvar::kFrameWidth * sizeof(vec3);
    vec3 *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBufferSize));

    RenderManager *world;
    checkCudaErrors(hipMalloc((void **)&world, sizeof(RenderManager)));

    point3 lookfrom(13,2,3);
    point3 lookat(0,0,0);
    vec3 vup(0,1,0);
    auto dist_to_focus = 10.0f;
    auto aperture = 0.1f;
    camera *devCamera, *hostCamera =
            new camera(lookfrom, lookat, vup, 20, globalvar::kAspectRatio, aperture, dist_to_focus, 0.0, 1.0);
    checkCudaErrors(hipMalloc((void **)&devCamera, sizeof(camera)));
    checkCudaErrors(hipMemcpy(devCamera, hostCamera, sizeof(camera), hipMemcpyHostToDevice));

    //generateWorld<<<1, 1>>>(world);

    dim3 blocksSpp(globalvar::kBlockX, globalvar::kBlockY, globalvar::kSpp);
    dim3 blocks(globalvar::kBlockX, globalvar::kBlockY);
    dim3 threads(globalvar::kThreadX, globalvar::kThreadY);
    //printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);

    hiprandState *devStates;
    checkCudaErrors(hipMalloc((void **)&devStates, frameBufferSize * sizeof(hiprandState)));
    srand(time(nullptr));
    int seed = rand();
    initRandom<<<blocks, threads>>>(devStates, globalvar::kFrameWidth, globalvar::kFrameHeight, seed);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    generateRandomWorld<<<1, 1>>>(world, devStates);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    GLFWwindow *window;
    myGlInit(&window, globalvar::kFrameWidth, globalvar::kFrameHeight);

    hipStream_t stream;
    hipEvent_t  event;

    checkCudaErrors(hipStreamCreateWithFlags(&stream,hipStreamDefault));
    checkCudaErrors(hipEventCreateWithFlags(&event,hipEventBlockingSync));

    Cuda2Gl* interop = new Cuda2Gl(2);

    int width, height;
    glfwGetFramebufferSize(window,&width,&height);
    interop->updateFrameSize(width,height);
    glfwSetWindowUserPointer(window,interop);
    while (!glfwWindowShouldClose(window))
    {
        pxl_glfw_fps(window);
        interop->getFrameSize(&width,&height);
        interop->mapGraphicResource(stream);
        cudaBindSurfaceToArray(surf, interop->getCudaArray());

        renderBySurface<<<blocks, threads>>>(globalvar::kFrameWidth, globalvar::kFrameHeight,
                            globalvar::kSpp, globalvar::kMaxDepth,
                            devCamera, world, devStates, goRender);
        //hipDeviceSynchronize();

        //if(ii++ > count) goRender = false;

        interop->unMapGraphicResource(stream);
        interop->blitFramebuffer();
        interop->swapBuffer();
        glfwSwapBuffers(window);
        glfwPollEvents(); // glfwWaitEvents();
    }
    delete interop;
    glfwDestroyWindow(window);
    glfwTerminate();
    hipDeviceReset();


//    printf("Start rendering!\n");
//    std::clock_t start = std::clock();
//    render<<<blocks, threads>>>(frameBuffer, globalvar::kFrameWidth, globalvar::kFrameHeight,
//                                globalvar::kSpp, globalvar::kMaxDepth,
//                                devCamera, world, devStates);
////    renderPerSpp<<<blocksSpp, threads>>>(frameBuffer, globalvar::kFrameWidth, globalvar::kFrameHeight,
////                                1.0f / globalvar::kSpp, globalvar::kMaxDepth,
////                                devCamera, world, devStates);
//    checkCudaErrors(hipGetLastError());
//    checkCudaErrors(hipDeviceSynchronize());
//    auto duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
//    std::cout<<"Time Cost: "<< duration <<'\n';
//    for(int row = 0; row < globalvar::kFrameHeight; row++){
//        for(int col = 0; col < globalvar::kFrameWidth; col++){
//            int curPixel = row * globalvar::kFrameWidth + col;
//            //std::cerr << frameBuffer[curPixel] << '\n';
//            png.saveColor(frameBuffer[curPixel], globalvar::kFrameHeight - row - 1, col);
//        }
//    }
//
//    png.write("../output2/3.png");
    clearWorld<<<1, 1>>>(world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(frameBuffer));
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(devStates));
    checkCudaErrors(hipFree(devCamera));
    hipDeviceReset();
    return 0;
}




void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow *window);

// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 600;


int draw(){
    //renderScene();
    // glfw: initialize and configure
    // ------------------------------
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif
    // glfw window creation
    // --------------------
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // glad: load all OpenGL function pointers
    // ---------------------------------------
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    // render loop
    // -----------
    while (!glfwWindowShouldClose(window))
    {
        // input
        // -----
        processInput(window);

        // render
        // ------
        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
        // -------------------------------------------------------------------------------
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // glfw: terminate, clearing all previously allocated GLFW resources.
    // ------------------------------------------------------------------
    glfwTerminate();
    return 0;
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow *window)
{
    if(glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}


